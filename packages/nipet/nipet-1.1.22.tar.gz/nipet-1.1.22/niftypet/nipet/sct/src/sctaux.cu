#include "hip/hip_runtime.h"
/*------------------------------------------------------------------------
Python extension for CUDA auxiliary routines used in 
voxel-driven scatter modelling (VSM)

author: Pawel Markiewicz
Copyrights: 2018
------------------------------------------------------------------------*/
#include <stdlib.h>
#include "sctaux.h"

void HandleError(hipError_t err, const char *file, int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
		exit(EXIT_FAILURE);
	}
}

//************ CHECK DEVICE MEMORY USAGE *********************
void getMemUse(Cnst Cnt) {
	size_t free_mem;
	size_t total_mem;
	HANDLE_ERROR(hipMemGetInfo(&free_mem, &total_mem));
	double free_db = (double)free_mem;
	double total_db = (double)total_mem;
	double used_db = total_db - free_db;
	if (Cnt.VERBOSE == 1) printf("\ni> current GPU memory usage: %7.2f/%7.2f [MB]\n", used_db / 1024.0 / 1024.0, total_db / 1024.0 / 1024.0);
}
//************************************************************


//==========================================================================================
//  S C A T T E R
//==========================================================================================

//------------- DEFINE A SUBSET OF CRYSTAL and THEIR CENTRES FOR SCATTER -------------------
scrsDEF def_scrs(short * isrng, float *crs, Cnst Cnt)
{

	scrsDEF d_scrsdef;
	float * scrs = (float*)malloc(3 * nCRS * sizeof(float));
	//indx of scatter crystals, ending with the total number
	int iscrs = 0;
	//counter for crystal period, SCRS_T
	int cntr = 0;

	for (int c = 0; c<nCRS; c++) {
		if (((c + 1) % 9) == 0) continue;
		cntr += 1;
		if (cntr == SCRS_T) {
			cntr = 0;
			scrs[3 * iscrs] = (float)c;
			scrs[3 * iscrs + 1] = 0.5*(crs[c] + crs[c + 2 * nCRS]);
			scrs[3 * iscrs + 2] = 0.5*(crs[c + nCRS] + crs[c + 3 * nCRS]);

			// printf("i> %d-th scatter crystal (%d): (x,y) = (%2.2f, %2.2f). \n", iscrs, c, scrs[3*iscrs+1], scrs[3*iscrs+2]);
			iscrs += 1;
		}
	}

	//scatter ring definitions
#ifdef WIN32
	float *h_scrcdefRng, *h_scrsdefCrs;
	HANDLE_ERROR(hipHostMalloc(&h_scrcdefRng, 2 * Cnt.NSRNG * sizeof(float)));
	float z = 0.5*(-Cnt.NRNG*Cnt.AXR + Cnt.AXR);
	for (int ir = 0; ir<Cnt.NSRNG; ir++) {
		h_scrcdefRng[2 * ir] = (float)isrng[ir];
		h_scrcdefRng[2 * ir + 1] = z + isrng[ir] * Cnt.AXR;
		if (Cnt.VERBOSE == 1) printf(">> [%d]: ring_i=%d, ring_z=%f\n", ir, (int)h_scrcdefRng[2 * ir], h_scrcdefRng[2 * ir + 1]);
	}
	HANDLE_ERROR(hipMalloc(&d_scrsdef.rng, 2 * Cnt.NSRNG * sizeof(float)));
	HANDLE_ERROR(hipMemcpy(d_scrsdef.rng, h_scrcdefRng, 2 * Cnt.NSRNG * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipHostFree(h_scrcdefRng));

	//transaxial crs to structure
	HANDLE_ERROR(hipHostMalloc(&h_scrsdefCrs, 3 * iscrs * sizeof(float)));
	for (int sc = 0; sc<iscrs; sc++) {
		h_scrsdefCrs[3 * sc] = scrs[3 * sc];
		h_scrsdefCrs[3 * sc + 1] = scrs[3 * sc + 1];
		h_scrsdefCrs[3 * sc + 2] = scrs[3 * sc + 2];
		if (Cnt.VERBOSE == 1) printf("i> %d-th scatter crystal (%d): (x,y) = (%2.2f, %2.2f). \n", sc, (int)h_scrsdefCrs[3 * sc], h_scrsdefCrs[3 * sc + 1], h_scrsdefCrs[3 * sc + 2]);
	}
	HANDLE_ERROR(hipMalloc(&d_scrsdef.crs, 3 * iscrs * sizeof(float)));
	HANDLE_ERROR(hipMemcpy(d_scrsdef.crs, h_scrsdefCrs, 3 * iscrs * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipHostFree(h_scrsdefCrs));

#else
	HANDLE_ERROR(hipMallocManaged(&d_scrsdef.rng, 2 * Cnt.NSRNG * sizeof(float)));
	float z = 0.5*(-Cnt.NRNG*Cnt.AXR + Cnt.AXR);
	for (int ir = 0; ir<Cnt.NSRNG; ir++) {
		d_scrsdef.rng[2 * ir] = (float)isrng[ir];
		d_scrsdef.rng[2 * ir + 1] = z + isrng[ir] * Cnt.AXR;
		if (Cnt.VERBOSE == 1) printf(">> [%d]: ring_i=%d, ring_z=%f\n", ir, (int)d_scrsdef.rng[2 * ir], d_scrsdef.rng[2 * ir + 1]);
	}

	//transaxial crs to structure
	HANDLE_ERROR(hipMallocManaged(&d_scrsdef.crs, 3 * iscrs * sizeof(float)));
	for (int sc = 0; sc<iscrs; sc++) {
		d_scrsdef.crs[3 * sc] = scrs[3 * sc];
		d_scrsdef.crs[3 * sc + 1] = scrs[3 * sc + 1];
		d_scrsdef.crs[3 * sc + 2] = scrs[3 * sc + 2];
		if (Cnt.VERBOSE == 1) printf("i> %d-th scatter crystal (%d): (x,y) = (%2.2f, %2.2f). \n", sc, (int)d_scrsdef.crs[3 * sc], d_scrsdef.crs[3 * sc + 1], d_scrsdef.crs[3 * sc + 2]);
	}
#endif


	d_scrsdef.nscrs = iscrs;
	d_scrsdef.nsrng = Cnt.NSRNG;
	Cnt.NSCRS = iscrs;

	free(scrs);

	return d_scrsdef;
}


//==========================================================================
//---------- get 3D scatter look up tables ---------------------------------
int * get_2DsctLUT(scrsDEF d_scrsdef, Cnst Cnt) {


	//crystals -> sino bin LUT
	short c0, c1;
	int * c2s = (int*)malloc(Cnt.NCRS*Cnt.NCRS * sizeof(int));
	for (int iw = 0; iw<Cnt.W; iw++) {
		for (int ia = 0; ia<Cnt.A; ia++) {
			c0 = floor(fmod(ia + .5*(Cnt.NCRS - 2 + Cnt.W / 2 - iw), Cnt.NCRS));
			c1 = floor(fmod(ia + .5*(2 * Cnt.NCRS - 2 - Cnt.W / 2 + iw), Cnt.NCRS));
			c2s[c0 + c1*Cnt.NCRS] = iw + ia*Cnt.W;//ia + iw*Cnt.A;
			c2s[c1 + c0*Cnt.NCRS] = iw + ia*Cnt.W;//ia + iw*Cnt.A;
		}
	}

	int *d_sct2aw;

#ifdef WIN32
	int *h_sct2aw;
	float *h_scrsdefCrs;
	HANDLE_ERROR(hipHostMalloc(&h_sct2aw, d_scrsdef.nscrs*d_scrsdef.nscrs / 2 * sizeof(int)));
	HANDLE_ERROR(hipMalloc(&d_sct2aw, d_scrsdef.nscrs*d_scrsdef.nscrs / 2 * sizeof(int)));
	//HANDLE_ERROR(hipMalloc(&d_scrsdef.crs, 3 * iscrs * sizeof(float)));

	HANDLE_ERROR(hipHostMalloc(&h_scrsdefCrs, 3 * d_scrsdef.nscrs * sizeof(float)));
	HANDLE_ERROR(hipMemcpy(h_scrsdefCrs, d_scrsdef.crs, 3 * d_scrsdef.nscrs * sizeof(float), hipMemcpyDeviceToHost));

	//printf("i> d_scrsdef: nrcs nsrng %d %d\n\n", d_scrsdef.nscrs, d_scrsdef.nsrng);


	//loop over unscattered crystals
	for (int uc = 0; uc<d_scrsdef.nscrs; uc++) {

		//loop over scatter crystals
		for (int i = 0; i<d_scrsdef.nscrs / 2; i++) {
			//scatter crystal based on the position of unscatter crystal <uc>
			int sc = (uc + d_scrsdef.nscrs / 4 + i) & (d_scrsdef.nscrs - 1);
			//sino linear index (full including the gaps)      
			h_sct2aw[d_scrsdef.nscrs / 2 * uc + i] = c2s[(int)h_scrsdefCrs[3 * uc] + Cnt.NCRS*(int)h_scrsdefCrs[3 * sc]];

			//scattered and unscattered crystal positions (used for determining +/- sino segments)
			float xs = h_scrsdefCrs[3 * sc + 1];
			float xu = h_scrsdefCrs[3 * uc + 1];

			if (xs>xu) { h_sct2aw[d_scrsdef.nscrs / 2 * uc + i] += (1 << 30); }

			// printf("uc = %d (c=%d, xu = %f), sc = %d (c=%d, xs = %f), iAW = %d\n",
			//        uc, (int)d_scrsdef.crs[3*uc], d_scrsdef.crs[3*uc+1],
			//        sc, (int)d_scrsdef.crs[3*sc], d_scrsdef.crs[3*sc+1],
			//        d_sct2aw[d_scrsdef.nscrs/2*uc + i] );
		}

	}
	HANDLE_ERROR(hipMemcpy(d_sct2aw, h_sct2aw, d_scrsdef.nscrs*d_scrsdef.nscrs / 2 * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipHostFree(h_sct2aw));
	HANDLE_ERROR(hipHostFree(h_scrsdefCrs));



#else

	HANDLE_ERROR(hipMallocManaged(&d_sct2aw, d_scrsdef.nscrs*d_scrsdef.nscrs / 2 * sizeof(int)));

	//loop over unscattered crystals
	for (int uc = 0; uc<d_scrsdef.nscrs; uc++) {

		//loop over scatter crystals
		for (int i = 0; i<d_scrsdef.nscrs / 2; i++) {
			//scatter crystal based on the position of unscatter crystal <uc>
			int sc = (uc + d_scrsdef.nscrs / 4 + i) & (d_scrsdef.nscrs - 1);
			//sino linear index (full including the gaps)      
			d_sct2aw[d_scrsdef.nscrs / 2 * uc + i] = c2s[(int)d_scrsdef.crs[3 * uc] + Cnt.NCRS*(int)d_scrsdef.crs[3 * sc]];

			//scattered and unscattered crystal positions (used for determining +/- sino segments)
			float xs = d_scrsdef.crs[3 * sc + 1];
			float xu = d_scrsdef.crs[3 * uc + 1];

			if (xs>xu) { d_sct2aw[d_scrsdef.nscrs / 2 * uc + i] += (1 << 30); }

			// printf("uc = %d (c=%d, xu = %f), sc = %d (c=%d, xs = %f), iAW = %d\n",
			//        uc, (int)d_scrsdef.crs[3*uc], d_scrsdef.crs[3*uc+1],
			//        sc, (int)d_scrsdef.crs[3*sc], d_scrsdef.crs[3*sc+1],
			//        d_sct2aw[d_scrsdef.nscrs/2*uc + i] );
		}

	}

#endif

	return d_sct2aw;
}



//---------------- Scatter crystals to sino bins -------------------------------------
snLUT get_scrs2sn(int nscrs, float *scrs, Cnst Cnt) {

	snLUT lut;

	//first the usual crystals -> sino bin
	short c1, c2;
	int * c2s = (int*)malloc(nCRS*nCRS * sizeof(int));
	for (int iw = 0; iw<NSBINS; iw++) {
		for (int ia = 0; ia<NSANGLES; ia++) {
			c1 = floor(fmod(ia + .5*(nCRS - 2 + NSBINS / 2 - iw), nCRS));
			c2 = floor(fmod(ia + .5*(2 * nCRS - 2 - NSBINS / 2 + iw), nCRS));
			c2s[c1 + c2*nCRS] = ia + iw*NSANGLES;
			c2s[c2 + c1*nCRS] = ia + iw*NSANGLES;
		}
	}

	lut.crs2sn = c2s;

	//===========================================
	//array of luts:
	//[0]: scatter results -> linear sino index
	//[1]: scatter results -> linear index of summed results (usually 2 results per sino bin)
	int *d_sct2sn;

#ifdef WIN32
	int *h_sct2sn;
	HANDLE_ERROR(hipHostMalloc(&h_sct2sn, nscrs*nscrs * sizeof(int)));
	HANDLE_ERROR(hipMalloc(&d_sct2sn, nscrs*nscrs * sizeof(int)));

	//for checking if sino bin was already accounted for
	int * chcksino = (int*)malloc(NSBINANG * sizeof(int));
	memset(chcksino, 0, NSBINANG * sizeof(int));

	int cnt = 0;

	//uc: unscattered photon crystal, sc: scattered photon crystal
	for (int uc = 0; uc<nscrs; uc++) {

		for (int i = 0; i<nscrs / 2; i++) {
			int sc = (uc + nscrs / 4 + i) & (nscrs - 1);
			//sino linear index
			int sn_i = c2s[(int)scrs[3 * uc] + nCRS*(int)scrs[3 * sc]];
			h_sct2sn[nscrs*uc + 2 * i] = sn_i << 1;

			if (chcksino[sn_i] == 0) {
				cnt += 1;
				chcksino[sn_i] = cnt;
				h_sct2sn[nscrs*uc + 2 * i + 1] = cnt - 1;
			}
			else
				h_sct2sn[nscrs*uc + 2 * i + 1] = chcksino[sn_i] - 1;

			//printf("uc = %d, sci = %d, sni = %d, cnt = %d\n", uc, i, sn_i, d_sct2sn[nscrs*uc + 2*i+1]);
		}

	}

	HANDLE_ERROR(hipMemcpy(d_sct2sn, h_sct2sn, nscrs*nscrs * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipHostFree(h_sct2sn));

#else

	HANDLE_ERROR(hipMallocManaged(&d_sct2sn, nscrs*nscrs * sizeof(int)));

	//for checking if sino bin was already accounted for
	int * chcksino = (int*)malloc(NSBINANG * sizeof(int));
	memset(chcksino, 0, NSBINANG * sizeof(int));

	int cnt = 0;

	//uc: unscattered photon crystal, sc: scattered photon crystal
	for (int uc = 0; uc<nscrs; uc++) {

		for (int i = 0; i<nscrs / 2; i++) {
			int sc = (uc + nscrs / 4 + i) & (nscrs - 1);
			//sino linear index
			int sn_i = c2s[(int)scrs[3 * uc] + nCRS*(int)scrs[3 * sc]];
			d_sct2sn[nscrs*uc + 2 * i] = sn_i << 1;

			if (chcksino[sn_i] == 0) {
				cnt += 1;
				chcksino[sn_i] = cnt;
				d_sct2sn[nscrs*uc + 2 * i + 1] = cnt - 1;
			}
			else
				d_sct2sn[nscrs*uc + 2 * i + 1] = chcksino[sn_i] - 1;

			//printf("uc = %d, sci = %d, sni = %d, cnt = %d\n", uc, i, sn_i, d_sct2sn[nscrs*uc + 2*i+1]);
		}

	}

#endif

	lut.sct2sn = d_sct2sn;
	lut.nsval = cnt;

	if (Cnt.VERBOSE == 1) printf("i> number of sino bins used in scatter sinogram: %d\n\n", cnt);


	return lut;
}
//----------------------------------------------------------------------------------





//============================================================================
//SCATTER RESULTS PROCESSING
//============================================================================

__constant__ short c_isrng[N_SRNG];

__global__ void d_sct2sn1(float *scts1,
	float *srslt,
	size_t offtof,
	int *sct2D_AW,
	short *offseg,
	int NBIN,
	int MRD)
{
	//scatter crystal index
	char ics = threadIdx.x;

	//scatter ring index
	char irs = threadIdx.y;

	//unscattered crystal index
	char icu = blockIdx.x;
	//unscattered crystal index
	char iru = blockIdx.y;



	//number of considered crystals and rings for scatter
	char nscrs = gridDim.x;
	char nsrng = gridDim.y;

	//scatter bin index for one scatter sino/plane
	short ssi = nscrs / 2 * icu + ics;
	//int iAW = sct2D_AW[ ssi ] & 0x3fffffff;
	bool pos = ((2 * (sct2D_AW[ssi] >> 30) - 1) * (irs - iru)) > 0;

	// ring difference index used for addressing the segment offset to obtain sino index in span-1
	unsigned short rd = __usad(c_isrng[irs], c_isrng[iru], 0);

	//if(rd<=MRD)
	{
		unsigned short rdi = (2 * rd - 1 * pos);
		unsigned short sni = offseg[rdi] + MIN(c_isrng[irs], c_isrng[iru]);

		atomicAdd(scts1 + sni*NBIN + ssi,
			srslt[offtof + iru * nscrs*nsrng*nscrs / 2 + icu * nsrng*nscrs / 2 + irs*nscrs / 2 + ics]);
	}
}

//~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
__global__ void d_sct_axinterp(float *sct3d,
	const float *scts1,
	const int4 *sctaxR,
	const float4 *sctaxW,
	const short *sn1_sn11,
	int NBIN,
	int NSN1,
	int SPN,
	int offtof)
{
	//scatter crystal index
	char ics = threadIdx.x;

	//unscattered crystal index
	char icu = 2 * threadIdx.y;

	//span-1 sino index
	short sni = blockIdx.x;

	float tmp1, tmp2;

	tmp1 = sctaxW[sni].x * scts1[NBIN*sctaxR[sni].x + icu*blockDim.x + ics] +
		sctaxW[sni].y * scts1[NBIN*sctaxR[sni].y + icu*blockDim.x + ics] +
		sctaxW[sni].z * scts1[NBIN*sctaxR[sni].z + icu*blockDim.x + ics] +
		sctaxW[sni].w * scts1[NBIN*sctaxR[sni].w + icu*blockDim.x + ics];

	//for the rest of the unscattered crystals (due to limited indexing of 1024 in a block)
	icu += 1;
	tmp2 = sctaxW[sni].x * scts1[NBIN*sctaxR[sni].x + icu*blockDim.x + ics] +
		sctaxW[sni].y * scts1[NBIN*sctaxR[sni].y + icu*blockDim.x + ics] +
		sctaxW[sni].z * scts1[NBIN*sctaxR[sni].z + icu*blockDim.x + ics] +
		sctaxW[sni].w * scts1[NBIN*sctaxR[sni].w + icu*blockDim.x + ics];


	//span-1 or span-11 scatter pre-sinogram interpolation
	if (SPN == 1) {
		sct3d[offtof + sni*NBIN + (icu - 1)*blockDim.x + ics] = tmp1;
		sct3d[offtof + sni*NBIN + icu*blockDim.x + ics] = tmp2;
	}
	else if (SPN == 11) {
		//only converting to span-11 when MRD<=60
		if (sni<NSN1) {
			short sni11 = sn1_sn11[sni];
			atomicAdd(sct3d + offtof + sni11*NBIN + (icu - 1)*blockDim.x + ics, tmp1);
			atomicAdd(sct3d + offtof + sni11*NBIN + icu*blockDim.x + ics, tmp2);
		}
	}

}


//=============================================================================
float * srslt2sino(float *d_srslt,
	int *d_sct2D_AW,
	scrsDEF d_scrsdef,
	int *sctaxR,
	float *sctaxW,
	short *offseg,
	short *isrng,
	short *sn1_rno,
	short *sn1_sn11,
	Cnst Cnt)
{

	//scatter pre-sino in span-1 (tmporary) 
	float *d_scts1;
	HANDLE_ERROR(hipMalloc(&d_scts1, Cnt.NSN64*d_scrsdef.nscrs*d_scrsdef.nscrs / 2 * sizeof(float)));


	//axially interpolated scatter pre-sino; full span-1 without MRD limit or span-11 with MRD=60
	float *d_sct3di;
	int tbins = 0;
	if (Cnt.SPN == 1)
		tbins = Cnt.NSN64*d_scrsdef.nscrs*d_scrsdef.nscrs / 2;
	//scatter pre-sino, span-11
	else if (Cnt.SPN == 11)
		tbins = Cnt.NSN11*d_scrsdef.nscrs*d_scrsdef.nscrs / 2;
	HANDLE_ERROR(hipMalloc(&d_sct3di, Cnt.TOFBINN*tbins * sizeof(float)));
	HANDLE_ERROR(hipMemset(d_sct3di, 0, Cnt.TOFBINN*tbins * sizeof(float)));

	//number of all scatter estimated values (sevn) for one TOF 3D sino
	int sevn = d_scrsdef.nsrng*d_scrsdef.nscrs*d_scrsdef.nsrng*d_scrsdef.nscrs / 2;

	//---- constants  
	int4 *d_sctaxR;
	HANDLE_ERROR(hipMalloc(&d_sctaxR, Cnt.NSN64 * sizeof(int4)));
	HANDLE_ERROR(hipMemcpy(d_sctaxR, sctaxR, Cnt.NSN64 * sizeof(int4), hipMemcpyHostToDevice));

	float4 *d_sctaxW;
	HANDLE_ERROR(hipMalloc(&d_sctaxW, Cnt.NSN64 * sizeof(float4)));
	HANDLE_ERROR(hipMemcpy(d_sctaxW, sctaxW, Cnt.NSN64 * sizeof(float4), hipMemcpyHostToDevice));

	short *d_offseg;
	HANDLE_ERROR(hipMalloc(&d_offseg, (Cnt.NSEG0 + 1) * sizeof(short)));
	HANDLE_ERROR(hipMemcpy(d_offseg, offseg, (Cnt.NSEG0 + 1) * sizeof(short), hipMemcpyHostToDevice));

	if (N_SRNG != Cnt.NSRNG) printf("e> Number of scatter rings is different in definistions from Python! <<<<<<<<<<<<<<<<<<< error \n");
	//---scatter ring indecies to constant memory (GPU)
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(c_isrng), isrng, Cnt.NSRNG * sizeof(short)));
	//---

	short2 *d_sn1_rno;
	HANDLE_ERROR(hipMalloc(&d_sn1_rno, Cnt.NSN1 * sizeof(short2)));
	HANDLE_ERROR(hipMemcpy(d_sn1_rno, sn1_rno, Cnt.NSN1 * sizeof(short2), hipMemcpyHostToDevice));

	short *d_sn1_sn11;
	HANDLE_ERROR(hipMalloc(&d_sn1_sn11, Cnt.NSN1 * sizeof(short)));
	HANDLE_ERROR(hipMemcpy(d_sn1_sn11, sn1_sn11, Cnt.NSN1 * sizeof(short), hipMemcpyHostToDevice));
	//----

	for (int i = 0; i<Cnt.TOFBINN; i++) {

		//offset for given TOF bin
		size_t offtof = i*sevn;

		//init to zeros
		HANDLE_ERROR(hipMemset(d_scts1, 0, Cnt.NSN64*d_scrsdef.nscrs*d_scrsdef.nscrs / 2 * sizeof(float)));


		if (Cnt.VERBOSE == 1) printf("i> 3D scatter results into span-1 pre-sino for TOF bin %d...", i);
		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start, 0);
		//<><><><><><><><><><><><><><><><><><><><><><><><><><><><><>
		dim3 grid(d_scrsdef.nscrs, d_scrsdef.nsrng, 1);
		dim3 block(d_scrsdef.nscrs / 2, d_scrsdef.nsrng, 1);
		d_sct2sn1 << < grid, block >> >(d_scts1,
			d_srslt,
			offtof,
			d_sct2D_AW,
			d_offseg,
			(int)(d_scrsdef.nscrs*d_scrsdef.nscrs / 2),
			Cnt.MRD);
		hipError_t err = hipGetLastError();
		if (err != hipSuccess) printf("Error in d_sct2sn1: %s\n", hipGetErrorString(err));
		//<><><><><><><><><><><><><><><><><><><><><><><><><><><><><>
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		float elapsedTime;
		hipEventElapsedTime(&elapsedTime, start, stop);
		hipEventDestroy(start);
		hipEventDestroy(stop);
		if (Cnt.VERBOSE == 1) printf("DONE in %fs.\n", 1e-3*elapsedTime);



		if (Cnt.VERBOSE == 1) printf("i> 3D scatter axial interpolation...");
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start, 0);
		//<><><><><><><><><><><><><><><><><><><><><><><><><><><><><>
		block.x = d_scrsdef.nscrs / 2;
		block.y = d_scrsdef.nscrs / 2;
		block.z = 1;
		grid.x = Cnt.NSN64;
		grid.y = 1;
		grid.z = 1;
		d_sct_axinterp << < grid, block >> >(d_sct3di,
			d_scts1,
			d_sctaxR,
			d_sctaxW,
			d_sn1_sn11,
			(int)(d_scrsdef.nscrs*d_scrsdef.nscrs / 2),
			Cnt.NSN1,
			Cnt.SPN,
			i*tbins);
		err = hipGetLastError();
		if (err != hipSuccess) printf("Error in d_sct_axinterp: %s\n", hipGetErrorString(err));
		//<><><><><><><><><><><><><><><><><><><><><><><><><><><><><>
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsedTime, start, stop);
		hipEventDestroy(start);
		hipEventDestroy(stop);
		if (Cnt.VERBOSE == 1) printf("DONE in %fs.\n", 1e-3*elapsedTime);

	}

	hipFree(d_scts1);

	return d_sct3di;
}






//===================================================================
//------ CREATE MASK BASED ON THRESHOLD (SCATTER EMISSION DATA)------------
iMSK get_imskEm(IMflt imvol, float thrshld, Cnst Cnt)
{

	// check which device is going to be used
	int dev_id;
	hipGetDevice(&dev_id);
	if (Cnt.VERBOSE == 1) printf("ic> using CUDA device #%d\n", dev_id);

	iMSK msk;
	int nvx = 0;

	for (int i = 0; i<(SSE_IMX*SSE_IMY*SSE_IMZ); i++) {
		if (imvol.im[i]>thrshld)  nvx++;
	}
	//------------------------------------------------------------------
	//create the mask thru indexes
	int *d_i2v, *d_v2i;

#ifdef WIN32
	int *h_i2v, *h_v2i;
	HANDLE_ERROR(hipHostMalloc(&h_i2v, nvx * sizeof(int)));
	HANDLE_ERROR(hipHostMalloc(&h_v2i, SSE_IMX*SSE_IMY*SSE_IMZ * sizeof(int)));

	HANDLE_ERROR(hipMalloc(&d_i2v, nvx * sizeof(int))); // does d_12v and its kind get freed???????????????????????????????????????????
	HANDLE_ERROR(hipMalloc(&d_v2i, SSE_IMX*SSE_IMY*SSE_IMZ * sizeof(int)));

	nvx = 0;
	for (int i = 0; i<(SSE_IMX*SSE_IMY*SSE_IMZ); i++) {
		//if not in the mask then set to -1
		h_v2i[i] = 0;
		//image-based TFOV
		if (imvol.im[i]>thrshld) {
			h_i2v[nvx] = i;
			h_v2i[i] = nvx;
			nvx++;
		}
	}

	HANDLE_ERROR(hipMemcpy(d_i2v, h_i2v, nvx * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_v2i, h_v2i, SSE_IMX*SSE_IMY*SSE_IMZ * sizeof(int), hipMemcpyHostToDevice));

	HANDLE_ERROR(hipHostFree(h_i2v));
	HANDLE_ERROR(hipHostFree(h_v2i));


#else
	//printf(">>>>> NVX:%d, THRESHOLD:%f\n", nvx, thrshld);
	HANDLE_ERROR(hipMallocManaged(&d_i2v, nvx * sizeof(int)));
	HANDLE_ERROR(hipMallocManaged(&d_v2i, SSE_IMX*SSE_IMY*SSE_IMZ * sizeof(int)));

	nvx = 0;
	for (int i = 0; i<(SSE_IMX*SSE_IMY*SSE_IMZ); i++) {
		//if not in the mask then set to -1
		d_v2i[i] = 0;
		//image-based TFOV
		if (imvol.im[i]>thrshld) {
			d_i2v[nvx] = i;
			d_v2i[i] = nvx;
			nvx++;
		}
	}

#endif

	if (Cnt.VERBOSE == 1) printf("i> number of voxel values greater than %3.2f is %d out of %d (ratio: %3.2f)\n", thrshld, nvx, SSE_IMX*SSE_IMY*SSE_IMZ, nvx / (float)(SSE_IMX*SSE_IMY*SSE_IMZ));
	msk.nvx = nvx;
	msk.i2v = d_i2v;
	msk.v2i = d_v2i;
	return msk;
}
//===================================================================

//===================================================================
//----------- CREATE MASK BASED ON MASK PROVIDED ----------------
iMSK get_imskMu(IMflt imvol, char *msk, Cnst Cnt)
{

	// check which device is going to be used
	int dev_id;
	hipGetDevice(&dev_id);
	if (Cnt.VERBOSE == 1) printf("ic> using CUDA device #%d\n", dev_id);

	int nvx = 0;
	for (int i = 0; i<(SS_IMX*SS_IMY*SS_IMZ); i++) {
		if (msk[i]>0)  nvx++;
	}
	//------------------------------------------------------------------
	//create the mask thru indecies
	int *d_i2v, *d_v2i;

#ifdef WIN32
	int *h_i2v, *h_v2i;
	HANDLE_ERROR(hipHostMalloc(&h_i2v, nvx * sizeof(int)));
	HANDLE_ERROR(hipHostMalloc(&h_v2i, SS_IMX*SS_IMY*SS_IMZ * sizeof(int)));

	HANDLE_ERROR(hipMalloc(&d_i2v, nvx * sizeof(int)));
	HANDLE_ERROR(hipMalloc(&d_v2i, SS_IMX*SS_IMY*SS_IMZ * sizeof(int)));

	nvx = 0;
	for (int i = 0; i<(SS_IMX*SS_IMY*SS_IMZ); i++) {
		//if not in the mask then set to -1
		h_v2i[i] = -1;
		//image-based TFOV
		if (msk[i]>0) {
			h_i2v[nvx] = i;
			h_v2i[i] = nvx;
			nvx++;
		}
	}

	HANDLE_ERROR(hipMemcpy(d_i2v, h_i2v, nvx * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_v2i, h_v2i, SS_IMX*SS_IMY*SS_IMZ * sizeof(int), hipMemcpyHostToDevice));

	HANDLE_ERROR(hipHostFree(h_i2v));
	HANDLE_ERROR(hipHostFree(h_v2i));

#else

	HANDLE_ERROR(hipMallocManaged(&d_i2v, nvx * sizeof(int)));
	HANDLE_ERROR(hipMallocManaged(&d_v2i, SS_IMX*SS_IMY*SS_IMZ * sizeof(int)));

	nvx = 0;
	for (int i = 0; i<(SS_IMX*SS_IMY*SS_IMZ); i++) {
		//if not in the mask then set to -1
		d_v2i[i] = -1;
		//image-based TFOV
		if (msk[i]>0) {
			d_i2v[nvx] = i;
			d_v2i[i] = nvx;
			nvx++;
		}
	}

#endif
	if (Cnt.VERBOSE == 1) printf("i> number of voxels within the mu-mask is %d out of %d (ratio: %3.2f)\n", nvx, SS_IMX*SS_IMY*SS_IMZ, nvx / (float)(SS_IMX*SS_IMY*SS_IMZ));
	iMSK mlut;
	mlut.nvx = nvx;
	mlut.i2v = d_i2v;
	mlut.v2i = d_v2i;
	return mlut;
}





