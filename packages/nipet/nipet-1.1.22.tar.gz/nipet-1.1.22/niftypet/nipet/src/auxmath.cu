#include "hip/hip_runtime.h"
/*------------------------------------------------------------------------
CUDA C extension for Python
This extension module provides additional functionality for list-mode data
processing, converting between data structures for image reconstruction.

author: Pawel Markiewicz
Copyrights: 2018
------------------------------------------------------------------------*/


#include "auxmath.h"

#define MTHREADS 512

//=============================================================================
__global__ void var(float * M1,
	float * M2,
	float * X,
	int b,
	size_t nele) {
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx<nele) {
		float delta = X[idx] - M1[idx];
		M1[idx] += delta / (b + 1);
		M2[idx] += delta*(X[idx] - M1[idx]);
	}
}
//=============================================================================
//=============================================================================
void var_online(float *M1, float *M2, float *X, int b, size_t nele)
{
	
	//do calculation of variance online using CUDA kernel <var>.
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	float *d_m1; HANDLE_ERROR(hipMalloc(&d_m1, nele * sizeof(float)));
	HANDLE_ERROR(hipMemcpy(d_m1, M1, nele * sizeof(float), hipMemcpyHostToDevice));
	float *d_m2; HANDLE_ERROR(hipMalloc(&d_m2, nele * sizeof(float)));
	HANDLE_ERROR(hipMemcpy(d_m2, M2, nele * sizeof(float), hipMemcpyHostToDevice));
	float *d_x; HANDLE_ERROR(hipMalloc(&d_x, nele * sizeof(float)));
	HANDLE_ERROR(hipMemcpy(d_x, X, nele * sizeof(float), hipMemcpyHostToDevice));


	int blcks = (nele + MTHREADS - 1) / MTHREADS;
	var << < blcks, MTHREADS >> >(d_m1, d_m2, d_x, b, nele);


	//copy M1 and M2 back to CPU memory
	HANDLE_ERROR(hipMemcpy(M1, d_m1, nele * sizeof(float), hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(M2, d_m2, nele * sizeof(float), hipMemcpyDeviceToHost));

	hipFree(d_m1);
	hipFree(d_m2);
	hipFree(d_x);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	printf("i> online variance calculation DONE in %fs.\n\n", 0.001*elapsedTime);
}
//=============================================================================




//===============================================================================
__global__ void d_remgaps(float * sng,
	const float * sn,
	const int * aw2li,
	const int snno)
{
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx<AW) {

		float input;

		for (int i = 0; i<snno; i++) {
			input = (float)sn[aw2li[idx] + i*NSANGLES*NSBINS];
			sng[i + idx*snno] = input;
		}
	}
}

//--------------------------------------------------------------------------------
void remove_gaps(float *sng,
	float *sino,
	int snno,
	int *aw2ali,
	Cnst Cnt)
{
	// check which device is going to be used
	int dev_id;
	hipGetDevice(&dev_id);
	if (Cnt.VERBOSE == 1) printf("ic> using CUDA device #%d\n", dev_id);

	int nthreads = 256;
	int blcks = ceil(AW / (float)nthreads);

	float *d_sng; HANDLE_ERROR(hipMalloc(&d_sng, AW*snno * sizeof(float)));
	HANDLE_ERROR(hipMemset(d_sng, 0, AW*snno * sizeof(float)));

	float *d_sino; HANDLE_ERROR(hipMalloc(&d_sino, NSBINS*NSANGLES*snno * sizeof(float)));
	HANDLE_ERROR(hipMemcpy(d_sino, sino, NSBINS*NSANGLES*snno * sizeof(float), hipMemcpyHostToDevice));

	int *d_aw2ali;
	HANDLE_ERROR(hipMalloc(&d_aw2ali, AW * sizeof(int)));
	HANDLE_ERROR(hipMemcpy(d_aw2ali, aw2ali, AW * sizeof(int), hipMemcpyHostToDevice));

	if (Cnt.VERBOSE == 1)
		printf("i> and removing the gaps and reordering sino for GPU...");
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	//============================================================================
	d_remgaps << <blcks, nthreads >> >(d_sng, d_sino, d_aw2ali, snno);
	//============================================================================
	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
		printf("Error: %s\n", hipGetErrorString(err));

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	if (Cnt.VERBOSE == 1)
		printf(" DONE in %fs\n", 0.001*elapsedTime);

	HANDLE_ERROR(hipMemcpy(sng, d_sng, AW*snno * sizeof(float), hipMemcpyDeviceToHost));

	hipFree(d_sng);
	hipFree(d_sino);
	hipFree(d_aw2ali);

	return;
}


//=============================================================================
__global__ void d_putgaps(float *sne7,
	float *snaw,
	int *aw2ali,
	const int snno)
{
	//sino index
	int sni = threadIdx.x + blockIdx.y*blockDim.x;

	//sino bin index
	int awi = blockIdx.x;

	if (sni<snno) {
		sne7[aw2ali[awi] * snno + sni] = snaw[awi*snno + sni];
	}
}
//=============================================================================

//=============================================================================
void put_gaps(float *sino,
	float *sng,
	int *aw2ali,
	Cnst Cnt)
{
	// check which device is going to be used
	int dev_id;
	hipGetDevice(&dev_id);
	if (Cnt.VERBOSE == 1) printf("ic> using CUDA device #%d\n", dev_id);

	//number of sinos
	int snno = -1;
	//number of blocks of threads
	dim3 zBpG(AW, 1, 1);

	if (Cnt.SPN == 11) {
		// number of blocks (y) for CUDA launch
		zBpG.y = 2;
		snno = NSINOS11;
	}
	else if (Cnt.SPN == 1) {
		// number of blocks (y) for CUDA launch
		zBpG.y = 8;
		// number of direct rings considered
		int nrng_c = Cnt.RNG_END - Cnt.RNG_STRT;
		snno = nrng_c*nrng_c;
		//correct for the max. ring difference in the full axial extent (don't use ring range (1,63) as for this case no correction) 
		if (nrng_c == 64)  snno -= 12;
	}
	else {
		printf("e> not span-1 nor span-11\n");
		return;
	}

	//printf("ci> number of sinograms to put gaps in: %d\n", snno); REMOVED AS SCREEN OUTPUT IS TOO MUCH

	float *d_sng;
	HANDLE_ERROR(hipMalloc(&d_sng, AW*snno * sizeof(float)));
	HANDLE_ERROR(hipMemcpy(d_sng, sng, AW*snno * sizeof(float), hipMemcpyHostToDevice));

	float *d_sino;
	HANDLE_ERROR(hipMalloc(&d_sino, NSBINS*NSANGLES*snno * sizeof(float)));
	HANDLE_ERROR(hipMemset(d_sino, 0, NSBINS*NSANGLES*snno * sizeof(float)));

	int *d_aw2ali;
	HANDLE_ERROR(hipMalloc(&d_aw2ali, AW * sizeof(int)));
	HANDLE_ERROR(hipMemcpy(d_aw2ali, aw2ali, AW * sizeof(int), hipMemcpyHostToDevice));

	if (Cnt.VERBOSE == 1)
		printf("ic> put gaps in and reorder sino...");
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	//<><><><><><><><><><><><><><><><><><><><><><><><><><><><><>
	d_putgaps << < zBpG, 64 * 14 >> >(d_sino,
		d_sng,
		d_aw2ali,
		snno);
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) printf("Error in d_sn11_sne7: %s\n", hipGetErrorString(err));
	//<><><><><><><><><><><><><><><><><><><><><><><><><><><><><>
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	if (Cnt.VERBOSE == 1)
		printf("DONE in %fs.\n", 0.001*elapsedTime);

	HANDLE_ERROR(hipMemcpy(sino, d_sino, NSBINS*NSANGLES*snno * sizeof(float), hipMemcpyDeviceToHost));

	hipFree(d_sng);
	hipFree(d_sino);
	hipFree(d_aw2ali);
	return;
}
