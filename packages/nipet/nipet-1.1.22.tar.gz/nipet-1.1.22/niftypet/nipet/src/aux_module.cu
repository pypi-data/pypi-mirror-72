#include "hip/hip_runtime.h"
/*------------------------------------------------------------------------
CUDA C extension for Python
This extension module provides auxiliary functionality for list-mode data
processing, generating look-up tables for image reconstruction.

author: Pawel Markiewicz
Copyrights: 2018
------------------------------------------------------------------------*/


#include <Python.h>
#include <stdlib.h>
#include <numpy/arrayobject.h>
#include "def.h"
#include "norm.h"
#include "scanner_0.h"
#include "auxmath.h"


//=== PYTHON STUFF ===

//--- Docstrings
static char module_docstring[] =
"Auxilary routines for the mMR.";
static char norm_docstring[] =
"Creates norm 3D sinograms from components provided in a file.";
static char txLUTs_docstring[] =
"transaxial (2D) look up tables.";
static char s11lut_docstring[] =
"span-1 to span-11 look up table.";
static char sne7_docstring[] =
"GPU span-11 results to span-11 in Siemens format.";
static char sn11_docstring[] =
"Siemens span-11 sino to span-11 GPU format.";

static char varon_docstring[] =
"Calculate vector variance online.";
//---

//--- Available functions
static PyObject *mmr_norm(PyObject *self, PyObject *args);
static PyObject *mmr_txlut(PyObject *self, PyObject *args);
static PyObject *mmr_span11LUT(PyObject *self, PyObject *args);
static PyObject *mmr_pgaps(PyObject *self, PyObject *args);
static PyObject *mmr_rgaps(PyObject *self, PyObject *args);
static PyObject *aux_varon(PyObject *self, PyObject *args);

/* Module specification */
static PyMethodDef module_methods[] = {
	{ "norm",   mmr_norm,   METH_VARARGS, norm_docstring },
	{ "txlut",  mmr_txlut,  METH_VARARGS, txLUTs_docstring },
	{ "s1s11",  mmr_span11LUT, METH_VARARGS, s11lut_docstring },
	{ "pgaps",  mmr_pgaps,  METH_VARARGS, sne7_docstring },
	{ "rgaps",  mmr_rgaps,  METH_VARARGS, sn11_docstring },
	{ "varon",  aux_varon,  METH_VARARGS, varon_docstring },

	{ NULL, NULL, 0, NULL }
};
//---

//--- Initialize the module
PyMODINIT_FUNC initmmr_auxe(void)  //it HAS to be init______ and then the name of the shared lib.
{
	PyObject *m = Py_InitModule3("mmr_auxe", module_methods, module_docstring);
	if (m == NULL)
		return;

	/* Load NumPy functionality. */
	import_array();
}
//==============================================================================






//==============================================================================
// N O R M A L I S A T I O N  (component based)
//------------------------------------------------------------------------------

static PyObject *mmr_norm(PyObject *self, PyObject *args)
{

	//Structure of constants
	Cnst Cnt;
	//Dictionary of scanner constants
	PyObject * o_mmrcnst;
	// structure of norm C arrays (defined in norm.h).
	NormCmp normc;
	// structure of axial LUTs in C arrays (defined in norm.h).
	axialLUT axLUT;

	//Output norm sino
	PyObject * o_sino;
	// normalisation component dictionary.
	PyObject * o_norm_cmp;
	// axial LUT dicionary. contains such LUTs: li2rno, li2sn, li2nos.
	PyObject * o_axLUT;
	// 2D sino index LUT (dead bisn are out).
	PyObject * o_aw2ali;
	// singles buckets for dead time correction
	PyObject * o_bckts;

	//^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^
	/* Parse the input tuple */
	if (!PyArg_ParseTuple(args, "OOOOOO", &o_sino, &o_norm_cmp, &o_bckts, &o_axLUT, &o_aw2ali, &o_mmrcnst))
		return NULL;
	//^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^

	/* Interpret the input objects as numpy arrays. */
	//norm components:
	PyObject* pd_geo = PyDict_GetItemString(o_norm_cmp, "geo");
	PyObject* pd_cinf = PyDict_GetItemString(o_norm_cmp, "cinf");
	PyObject* pd_ceff = PyDict_GetItemString(o_norm_cmp, "ceff");
	PyObject* pd_axe1 = PyDict_GetItemString(o_norm_cmp, "axe1");
	PyObject* pd_dtp = PyDict_GetItemString(o_norm_cmp, "dtp");
	PyObject* pd_dtnp = PyDict_GetItemString(o_norm_cmp, "dtnp");
	PyObject* pd_dtc = PyDict_GetItemString(o_norm_cmp, "dtc");
	PyObject* pd_axe2 = PyDict_GetItemString(o_norm_cmp, "axe2");
	PyObject* pd_axf1 = PyDict_GetItemString(o_norm_cmp, "axf1");
	//axial LUTs:
	PyObject* pd_li2rno = PyDict_GetItemString(o_axLUT, "li2rno");
	PyObject* pd_li2sn = PyDict_GetItemString(o_axLUT, "li2sn");
	PyObject* pd_li2nos = PyDict_GetItemString(o_axLUT, "li2nos");
	PyObject* pd_sn1sn11 = PyDict_GetItemString(o_axLUT, "sn1_sn11");
	PyObject* pd_sn1rno = PyDict_GetItemString(o_axLUT, "sn1_rno");
	PyObject* pd_sn1sn11no = PyDict_GetItemString(o_axLUT, "sn1_sn11no");

	PyObject* pd_span = PyDict_GetItemString(o_mmrcnst, "SPN");
	Cnt.SPN = (int)PyInt_AsLong(pd_span);
	PyObject* pd_verbose = PyDict_GetItemString(o_mmrcnst, "VERBOSE");
	Cnt.VERBOSE = (bool)PyInt_AS_LONG(pd_verbose);
	PyObject* pd_devid = PyDict_GetItemString(o_mmrcnst, "DEVID");
	Cnt.DEVID = (char)PyInt_AS_LONG(pd_devid);

	//get the output sino
	PyObject* p_sino = PyArray_FROM_OTF(o_sino, NPY_FLOAT32, NPY_IN_ARRAY);

	//-- get the arrays from the dictionaries
	//norm components
	PyObject *p_geo = PyArray_FROM_OTF(pd_geo, NPY_FLOAT32, NPY_IN_ARRAY);
	PyObject *p_cinf = PyArray_FROM_OTF(pd_cinf, NPY_FLOAT32, NPY_IN_ARRAY);
	PyObject *p_ceff = PyArray_FROM_OTF(pd_ceff, NPY_FLOAT32, NPY_IN_ARRAY);
	PyObject *p_axe1 = PyArray_FROM_OTF(pd_axe1, NPY_FLOAT32, NPY_IN_ARRAY);
	PyObject *p_dtp = PyArray_FROM_OTF(pd_dtp, NPY_FLOAT32, NPY_IN_ARRAY);
	PyObject *p_dtnp = PyArray_FROM_OTF(pd_dtnp, NPY_FLOAT32, NPY_IN_ARRAY);
	PyObject *p_dtc = PyArray_FROM_OTF(pd_dtc, NPY_FLOAT32, NPY_IN_ARRAY);
	PyObject *p_axe2 = PyArray_FROM_OTF(pd_axe2, NPY_FLOAT32, NPY_IN_ARRAY);
	PyObject *p_axf1 = PyArray_FROM_OTF(pd_axf1, NPY_FLOAT32, NPY_IN_ARRAY);
	//then axLUTs
	PyObject *p_li2rno = PyArray_FROM_OTF(pd_li2rno, NPY_INT32, NPY_IN_ARRAY);
	PyObject *p_li2sn = PyArray_FROM_OTF(pd_li2sn, NPY_INT32, NPY_IN_ARRAY);
	PyObject *p_li2nos = PyArray_FROM_OTF(pd_li2nos, NPY_INT32, NPY_IN_ARRAY);
	PyObject *p_sn1sn11 = PyArray_FROM_OTF(pd_sn1sn11, NPY_INT16, NPY_IN_ARRAY);
	PyObject *p_sn1rno = PyArray_FROM_OTF(pd_sn1rno, NPY_INT16, NPY_IN_ARRAY);
	PyObject *p_sn1sn11no = PyArray_FROM_OTF(pd_sn1sn11no, NPY_INT8, NPY_IN_ARRAY);

	//2D sino index LUT:
	PyObject *p_aw2ali = PyArray_FROM_OTF(o_aw2ali, NPY_INT32, NPY_IN_ARRAY);
	// single bucktes:
	PyObject *p_bckts = PyArray_FROM_OTF(o_bckts, NPY_INT32, NPY_IN_ARRAY);
	//--

	/* If that didn't work, throw an exception. */
	if (p_geo == NULL || p_cinf == NULL || p_ceff == NULL || p_axe1 == NULL ||
		p_dtp == NULL || p_dtnp == NULL || p_dtc == NULL || p_axe2 == NULL ||
		p_axf1 == NULL || p_li2rno == NULL || p_li2sn == NULL || p_li2nos == NULL ||
		p_aw2ali == NULL || p_sn1sn11 == NULL || p_sn1rno == NULL || p_sn1sn11no == NULL ||
		p_sino == NULL)
	{
		Py_XDECREF(p_geo);
		Py_XDECREF(p_cinf);
		Py_XDECREF(p_ceff);
		Py_XDECREF(p_axe1);
		Py_XDECREF(p_dtp);
		Py_XDECREF(p_dtnp);
		Py_XDECREF(p_dtc);
		Py_XDECREF(p_axe2);
		Py_XDECREF(p_axf1);
		//axLUTs
		Py_XDECREF(p_li2rno);
		Py_XDECREF(p_li2sn);
		Py_XDECREF(p_li2nos);
		Py_XDECREF(p_sn1sn11);
		Py_XDECREF(p_sn1rno);
		Py_XDECREF(p_sn1sn11no);
		//2D sino LUT
		Py_XDECREF(p_aw2ali);
		//singles buckets
		Py_XDECREF(p_bckts);
		//output sino
		Py_XDECREF(p_sino);
		return NULL;
	}

	//-- get the pointers to the data as C-types
	//norm components
	normc.geo = (float*)PyArray_DATA(p_geo);
	normc.cinf = (float*)PyArray_DATA(p_cinf);
	normc.ceff = (float*)PyArray_DATA(p_ceff);
	normc.axe1 = (float*)PyArray_DATA(p_axe1);
	normc.dtp = (float*)PyArray_DATA(p_dtp);
	normc.dtnp = (float*)PyArray_DATA(p_dtnp);
	normc.dtc = (float*)PyArray_DATA(p_dtc);
	normc.axe2 = (float*)PyArray_DATA(p_axe2);
	normc.axf1 = (float*)PyArray_DATA(p_axf1);
	//axLUTs
	axLUT.li2rno = (int*)PyArray_DATA(p_li2rno);
	axLUT.li2sn = (int*)PyArray_DATA(p_li2sn);
	axLUT.li2nos = (int*)PyArray_DATA(p_li2nos);
	axLUT.sn1_sn11 = (short*)PyArray_DATA(p_sn1sn11);
	axLUT.sn1_rno = (short*)PyArray_DATA(p_sn1rno);
	axLUT.sn1_sn11no = (char*)PyArray_DATA(p_sn1sn11no);

	//2D sino index LUT
	int * aw2ali = (int*)PyArray_DATA(p_aw2ali);
	//singles bucktes
	int * bckts = (int*)PyArray_DATA(p_bckts);

	//--- Array size
	int Naw = (int)PyArray_DIM(p_aw2ali, 0);
	if (AW != Naw) printf("\ne> number of active bins is inconsitent !!! <<------------------<<<<<\n");

	//output sino
	float *sino = (float *)PyArray_DATA(p_sino);

	//norm components
	normc.ngeo[0] = (int)PyArray_DIM(p_geo, 0);
	normc.ngeo[1] = (int)PyArray_DIM(p_geo, 1);
	normc.ncinf[0] = (int)PyArray_DIM(p_cinf, 0);
	normc.ncinf[1] = (int)PyArray_DIM(p_cinf, 1);
	normc.nceff[0] = (int)PyArray_DIM(p_ceff, 0);
	normc.nceff[1] = (int)PyArray_DIM(p_ceff, 1);
	normc.naxe = (int)PyArray_DIM(p_axe1, 0);
	normc.nrdt = (int)PyArray_DIM(p_dtp, 0);
	normc.ncdt = (int)PyArray_DIM(p_dtc, 0);
	//axial LUTs:
	axLUT.Nli2rno[0] = (int)PyArray_DIM(p_li2rno, 0);
	axLUT.Nli2rno[1] = (int)PyArray_DIM(p_li2rno, 1);
	axLUT.Nli2sn[0] = (int)PyArray_DIM(p_li2sn, 0);
	axLUT.Nli2sn[1] = (int)PyArray_DIM(p_li2sn, 1);
	axLUT.Nli2nos = (int)PyArray_DIM(p_li2nos, 0);

	// sets the device on which to calculate
	hipSetDevice(Cnt.DEVID);

	//<><><><><><><><><><> Call the CUDA stuff now
	norm_from_components(sino, normc, axLUT, aw2ali, bckts, Cnt);
	//<><><><><><><><><><>

	//-- Clear up
	//norm components
	Py_DECREF(p_geo);
	Py_DECREF(p_cinf);
	Py_DECREF(p_ceff);
	Py_DECREF(p_axe1);
	Py_DECREF(p_dtp);
	Py_DECREF(p_dtnp);
	Py_DECREF(p_dtc);
	Py_DECREF(p_axe2);
	//axLUT 
	Py_DECREF(p_li2rno);
	Py_DECREF(p_li2sn);
	Py_DECREF(p_li2nos);
	//2D sino index LUT
	Py_DECREF(p_aw2ali);
	//singles buckets
	Py_DECREF(p_bckts);
	//output sino
	Py_DECREF(p_sino);

	Py_INCREF(Py_None);
	return Py_None;

}





//======================================================================================
// E X T R A S
//--------------------------------------------------------------------------------------

//GET TRANSAXIAL LUTs
static PyObject *mmr_txlut(PyObject *self, PyObject *args) {
	//Dictionary of scanner constants
	PyObject * o_mmrcnst;

	//Structure of constants
	Cnst Cnt;

	//structure of transaxial LUTs
	txLUTs txluts;

	//^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^
	/* Parse the input tuple */
	if (!PyArg_ParseTuple(args, "O", &o_mmrcnst))
		return NULL;
	//^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^

	/* Interpret the input objects as... */
	PyObject* pd_A = PyDict_GetItemString(o_mmrcnst, "NSANGLES");
	Cnt.A = (int)PyInt_AsLong(pd_A);
	PyObject* pd_W = PyDict_GetItemString(o_mmrcnst, "NSBINS");
	Cnt.W = (int)PyInt_AsLong(pd_W);
	PyObject* pd_NSN1 = PyDict_GetItemString(o_mmrcnst, "NSN1");
	Cnt.NSN1 = (int)PyInt_AS_LONG(pd_NSN1);
	PyObject* pd_NSN11 = PyDict_GetItemString(o_mmrcnst, "NSN11");
	Cnt.NSN11 = (int)PyInt_AS_LONG(pd_NSN11);
	PyObject* pd_NRNG = PyDict_GetItemString(o_mmrcnst, "NRNG");
	Cnt.NRNG = (int)PyInt_AS_LONG(pd_NRNG);
	PyObject* pd_NCRS = PyDict_GetItemString(o_mmrcnst, "NCRS");
	Cnt.NCRS = (int)PyInt_AS_LONG(pd_NCRS);
	PyObject* pd_NCRSR = PyDict_GetItemString(o_mmrcnst, "NCRSR");
	Cnt.NCRSR = (int)PyInt_AS_LONG(pd_NCRSR);
	PyObject* pd_span = PyDict_GetItemString(o_mmrcnst, "SPN");
	Cnt.SPN = (int)PyInt_AS_LONG(pd_span);
	PyObject* pd_tgap = PyDict_GetItemString(o_mmrcnst, "TGAP");
	Cnt.TGAP = (int)PyInt_AS_LONG(pd_tgap);
	PyObject* pd_offgap = PyDict_GetItemString(o_mmrcnst, "OFFGAP");
	Cnt.OFFGAP = (int)PyInt_AS_LONG(pd_offgap);
	PyObject* pd_verbose = PyDict_GetItemString(o_mmrcnst, "VERBOSE");
	Cnt.VERBOSE = (bool)PyInt_AS_LONG(pd_verbose);

	txluts = get_txlut(Cnt);



	//---GET results out into Python tuples
	//sino to crystals (3 LUTs)
	npy_intp dims[2];
	dims[0] = Cnt.A*Cnt.W;
	dims[1] = 2;
	PyArrayObject *o_s2cF = (PyArrayObject *)PyArray_SimpleNewFromData(2, dims, NPY_INT16, txluts.s2cF);

	dims[0] = Cnt.NCRS;
	dims[1] = Cnt.NCRS;
	PyArrayObject *o_c2sF = (PyArrayObject *)PyArray_SimpleNewFromData(2, dims, NPY_INT32, txluts.c2sF);

	dims[0] = Cnt.NCRSR;
	dims[1] = Cnt.NCRSR;
	PyArrayObject *o_cr2s = (PyArrayObject *)PyArray_SimpleNewFromData(2, dims, NPY_INT32, txluts.cr2s);

	dims[0] = txluts.naw;
	dims[1] = 2;
	PyArrayObject *o_s2c = (PyArrayObject *)PyArray_SimpleNewFromData(2, dims, NPY_INT16, txluts.s2c);

	dims[0] = txluts.naw;
	dims[1] = 2;
	PyArrayObject *o_s2cr = (PyArrayObject *)PyArray_SimpleNewFromData(2, dims, NPY_INT16, txluts.s2cr);

	PyObject *tuple_s2c = PyTuple_New(5);
	PyTuple_SetItem(tuple_s2c, 0, PyArray_Return(o_s2cF));
	PyTuple_SetItem(tuple_s2c, 1, PyArray_Return(o_s2c));
	PyTuple_SetItem(tuple_s2c, 2, PyArray_Return(o_s2cr));
	PyTuple_SetItem(tuple_s2c, 3, PyArray_Return(o_c2sF));
	PyTuple_SetItem(tuple_s2c, 4, PyArray_Return(o_cr2s));

	//crystal index to active crystal index (avoiding dead crystal gaps)
	dims[0] = Cnt.NCRS;
	PyArrayObject *o_crsr = (PyArrayObject *)PyArray_SimpleNewFromData(1, dims, NPY_INT16, txluts.crsr);

	//linear 2D sino index to angle and bin sino idecies
	dims[0] = txluts.naw;
	dims[1] = 2;
	PyArrayObject *o_aw2sn = (PyArrayObject *)PyArray_SimpleNewFromData(2, dims, NPY_INT16, txluts.aw2sn);
	PyArrayObject *o_aw2ali = (PyArrayObject *)PyArray_SimpleNewFromData(1, dims, NPY_INT32, txluts.aw2ali);

	//crystal in coincidence (used for randoms estimation)
	dims[0] = Cnt.NCRSR;
	dims[1] = Cnt.NCRSR;
	PyArrayObject *o_cij = (PyArrayObject *)PyArray_SimpleNewFromData(2, dims, NPY_INT8, txluts.cij);

	//2D sino mask with 1's used to denote active bins. 
	dims[0] = Cnt.W;
	dims[1] = Cnt.A;
	PyArrayObject *o_msino = (PyArrayObject *)PyArray_SimpleNewFromData(2, dims, NPY_INT8, txluts.msino);


	//gather all together
	PyObject *tuple_out = PyTuple_New(7);
	PyTuple_SetItem(tuple_out, 0, Py_BuildValue("i", txluts.naw));
	PyTuple_SetItem(tuple_out, 1, tuple_s2c);
	PyTuple_SetItem(tuple_out, 2, PyArray_Return(o_crsr));
	PyTuple_SetItem(tuple_out, 3, PyArray_Return(o_cij));
	PyTuple_SetItem(tuple_out, 4, PyArray_Return(o_aw2sn));
	PyTuple_SetItem(tuple_out, 5, PyArray_Return(o_aw2ali));
	PyTuple_SetItem(tuple_out, 6, PyArray_Return(o_msino));

	return tuple_out;
}




//====================================================================================================
static PyObject *mmr_pgaps(PyObject *self, PyObject *args) {

	//output sino
	PyObject * o_sino;

	// transaxial LUT dictionary (e.g., 2D sino where dead bins are out).
	PyObject * o_txLUT;

	//Dictionary of scanner constants
	PyObject * o_mmrcnst;

	//GPU input sino in span-11
	PyObject * o_sng;

	//Structure of constants
	Cnst Cnt;

	//^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^
	/* Parse the input tuple */
	if (!PyArg_ParseTuple(args, "OOOO", &o_sino, &o_sng, &o_txLUT, &o_mmrcnst))
		return NULL;
	//^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^


	/* Interpret the input objects as... */
	PyObject* pd_NSN11 = PyDict_GetItemString(o_mmrcnst, "NSN11");
	Cnt.NSN11 = (int)PyInt_AS_LONG(pd_NSN11);
	PyObject* pd_A = PyDict_GetItemString(o_mmrcnst, "NSANGLES");
	Cnt.A = (int)PyInt_AsLong(pd_A);
	PyObject* pd_W = PyDict_GetItemString(o_mmrcnst, "NSBINS");
	Cnt.W = (int)PyInt_AsLong(pd_W);
	PyObject* pd_SPN = PyDict_GetItemString(o_mmrcnst, "SPN");
	Cnt.SPN = (int)PyInt_AS_LONG(pd_SPN);
	PyObject* pd_verbose = PyDict_GetItemString(o_mmrcnst, "VERBOSE");
	Cnt.VERBOSE = (bool)PyInt_AS_LONG(pd_verbose);
	PyObject* pd_devid = PyDict_GetItemString(o_mmrcnst, "DEVID");
	Cnt.DEVID = (char)PyInt_AS_LONG(pd_devid);

	PyObject* pd_rngstrt = PyDict_GetItemString(o_mmrcnst, "RNG_STRT");
	PyObject* pd_rngend = PyDict_GetItemString(o_mmrcnst, "RNG_END");
	Cnt.RNG_STRT = (char)PyInt_AS_LONG(pd_rngstrt);
	Cnt.RNG_END = (char)PyInt_AS_LONG(pd_rngend);

	//GPU 2D linear sino index into Siemens sino index LUT
	PyObject* pd_aw2ali = PyDict_GetItemString(o_txLUT, "aw2ali");

	//GPU input sino and the above 2D LUT
	PyObject *p_sng = PyArray_FROM_OTF(o_sng, NPY_FLOAT32, NPY_IN_ARRAY);
	PyObject *p_aw2ali = PyArray_FROM_OTF(pd_aw2ali, NPY_INT32, NPY_IN_ARRAY);

	//output sino
	PyObject *p_sino = PyArray_FROM_OTF(o_sino, NPY_FLOAT32, NPY_IN_ARRAY);

	if (p_sng == NULL || p_aw2ali == NULL || p_sino == NULL) {
		Py_XDECREF(p_aw2ali);
		Py_XDECREF(p_sng);
		Py_XDECREF(p_sino);
	}

	int *aw2ali = (int*)PyArray_DATA(p_aw2ali);
	float *sng = (float*)PyArray_DATA(p_sng);
	//output sino
	float *sino = (float*)PyArray_DATA(p_sino);

	// sets the device on which to calculate
	hipSetDevice(Cnt.DEVID);

	//<><><><><><><><><><><><><><><><><><><><><><>
	//Run the conversion to sinos with gaps
	put_gaps(sino, sng, aw2ali, Cnt);
	//<><><><><><><><><><><><><><><><><><><><><><>

	//Clean up
	Py_DECREF(p_aw2ali);
	Py_DECREF(p_sng);
	Py_DECREF(p_sino);

	Py_INCREF(Py_None);
	return Py_None;
}


//====================================================================================================
static PyObject *mmr_rgaps(PyObject *self, PyObject *args) {

	//output sino with gaps removed
	PyObject * o_sng;

	// transaxial LUT dictionary (e.g., 2D sino where dead bins are out).
	PyObject * o_txLUT;

	//Dictionary of scanner constants
	PyObject * o_mmrcnst;

	//input sino to be reformated with gaps removed
	PyObject * o_sino;

	//Structure of constants
	Cnst Cnt;

	//^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^
	/* Parse the input tuple */
	if (!PyArg_ParseTuple(args, "OOOO", &o_sng, &o_sino, &o_txLUT, &o_mmrcnst))
		return NULL;
	//^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^

	/* Interpret the input objects as... */
	PyObject* pd_NSN11 = PyDict_GetItemString(o_mmrcnst, "NSN11");
	Cnt.NSN11 = (int)PyInt_AS_LONG(pd_NSN11);
	PyObject* pd_NSN1 = PyDict_GetItemString(o_mmrcnst, "NSN1");
	Cnt.NSN1 = (int)PyInt_AS_LONG(pd_NSN1);
	PyObject* pd_A = PyDict_GetItemString(o_mmrcnst, "NSANGLES");
	Cnt.A = (int)PyInt_AsLong(pd_A);
	PyObject* pd_W = PyDict_GetItemString(o_mmrcnst, "NSBINS");
	Cnt.W = (int)PyInt_AsLong(pd_W);
	PyObject* pd_SPN = PyDict_GetItemString(o_mmrcnst, "SPN");
	Cnt.SPN = (int)PyInt_AS_LONG(pd_SPN);
	PyObject* pd_verbose = PyDict_GetItemString(o_mmrcnst, "VERBOSE");
	Cnt.VERBOSE = (bool)PyInt_AS_LONG(pd_verbose);
	PyObject* pd_devid = PyDict_GetItemString(o_mmrcnst, "DEVID");
	Cnt.DEVID = (char)PyInt_AS_LONG(pd_devid);

	//GPU 2D linear sino index into Siemens sino index LUT
	PyObject* pd_aw2ali = PyDict_GetItemString(o_txLUT, "aw2ali");

	//input sino and the above 2D LUT
	PyObject *p_sino = PyArray_FROM_OTF(o_sino, NPY_FLOAT32, NPY_IN_ARRAY);
	PyObject *p_aw2ali = PyArray_FROM_OTF(pd_aw2ali, NPY_INT32, NPY_IN_ARRAY);

	// number of sinogram from the shape of the sino (can be any number especially when using reduced ring number)
	int snno = (int)PyArray_DIM(p_sino, 0);

	//output sino
	PyObject *p_sng = PyArray_FROM_OTF(o_sng, NPY_FLOAT32, NPY_IN_ARRAY);

	if (p_sino == NULL || p_aw2ali == NULL || p_sino == NULL) {
		Py_XDECREF(p_aw2ali);
		Py_XDECREF(p_sino);
		Py_XDECREF(p_sng);
	}

	int *aw2ali = (int*)PyArray_DATA(p_aw2ali);
	float *sino = (float*)PyArray_DATA(p_sino);
	float *sng = (float*)PyArray_DATA(p_sng);

	// sets the device on which to calculate
	hipSetDevice(Cnt.DEVID);

	//<><><><><><><><><><><><><><><><><><><><><><>
	//Run the conversion to GPU sinos
	remove_gaps(sng, sino, snno, aw2ali, Cnt);
	//<><><><><><><><><><><><><><><><><><><><><><>

	//Clean up
	Py_DECREF(p_aw2ali);
	Py_DECREF(p_sino);
	Py_DECREF(p_sng);

	Py_INCREF(Py_None);
	return Py_None;

}



//====================================================================================================
static PyObject *mmr_span11LUT(PyObject *self, PyObject *args) {
	//Dictionary of scanner constants
	PyObject * o_mmrcnst;

	//Structure of constants
	Cnst Cnt;

	//^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^
	/* Parse the input tuple */
	if (!PyArg_ParseTuple(args, "O", &o_mmrcnst))
		return NULL;
	//^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^

	/* Interpret the input objects as... */
	PyObject* pd_Naw = PyDict_GetItemString(o_mmrcnst, "Naw");
	Cnt.aw = (int)PyInt_AS_LONG(pd_Naw);
	PyObject* pd_NSN1 = PyDict_GetItemString(o_mmrcnst, "NSN1");
	Cnt.NSN1 = (int)PyInt_AS_LONG(pd_NSN1);
	PyObject* pd_NSN11 = PyDict_GetItemString(o_mmrcnst, "NSN11");
	Cnt.NSN11 = (int)PyInt_AS_LONG(pd_NSN11);
	PyObject* pd_NRNG = PyDict_GetItemString(o_mmrcnst, "NRNG");
	Cnt.NRNG = (int)PyInt_AS_LONG(pd_NRNG);


	span11LUT span11 = span1_span11(Cnt);

	npy_intp dims[2];
	dims[0] = Cnt.NSN1;
	PyArrayObject *s1s11_out = (PyArrayObject *)PyArray_SimpleNewFromData(1, dims, NPY_INT16, span11.li2s11);
	dims[0] = Cnt.NSN11;
	PyArrayObject *s1nos_out = (PyArrayObject *)PyArray_SimpleNewFromData(1, dims, NPY_INT8, span11.NSinos);

	PyObject *o_out = PyTuple_New(2);
	PyTuple_SetItem(o_out, 0, PyArray_Return(s1s11_out));
	PyTuple_SetItem(o_out, 1, PyArray_Return(s1nos_out));


	return o_out;
}



//====================================================================================================
static PyObject *aux_varon(PyObject *self, PyObject *args) {

	// M1 (mean) vector 
	PyObject * o_m1;
	// M2 (variance) vector 
	PyObject * o_m2;
	//input of instance data X
	PyObject * o_x;
	//Dictionary of scanner constants
	PyObject * o_mmrcnst;

	//Structure of constants
	Cnst Cnt;
	//realisation number
	int b;

	//^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^
	/* Parse the input tuple */
	if (!PyArg_ParseTuple(args, "OOOiO", &o_m1, &o_m2, &o_x, &b, &o_mmrcnst))
		return NULL;
	//^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^

	PyObject* pd_verbose = PyDict_GetItemString(o_mmrcnst, "VERBOSE");
	Cnt.VERBOSE = (bool)PyInt_AS_LONG(pd_verbose);
	PyObject* pd_devid = PyDict_GetItemString(o_mmrcnst, "DEVID");
	Cnt.DEVID = (char)PyInt_AS_LONG(pd_devid);

	//input sino and the above 2D LUT
	PyObject *p_m1 = PyArray_FROM_OTF(o_m1, NPY_FLOAT32, NPY_IN_ARRAY);
	PyObject *p_m2 = PyArray_FROM_OTF(o_m2, NPY_FLOAT32, NPY_IN_ARRAY);
	PyObject *p_x = PyArray_FROM_OTF(o_x, NPY_FLOAT32, NPY_IN_ARRAY);

	if (p_m1 == NULL || p_m2 == NULL || p_x == NULL) {
		Py_XDECREF(p_m1);
		Py_XDECREF(p_m2);
		Py_XDECREF(p_x);
	}

	float *m1 = (float*)PyArray_DATA(p_m1);
	float *m2 = (float*)PyArray_DATA(p_m2);
	float *x = (float*)PyArray_DATA(p_x);
	int  ndim = PyArray_NDIM(p_x);
	size_t nele = 1;
	for (int i = 0; i<ndim; i++) {
		nele *= PyArray_DIM(p_x, i);
	}

	printf("i> number of elements in data array: %lu\n", nele);

	// sets the device on which to calculate
	hipSetDevice(Cnt.DEVID);

	//<><><><><><><><><><><><><><><><><><><><><><>
	//Update variance online (M1, M2) using data instance X
	var_online(m1, m2, x, b, nele);
	//<><><><><><><><><><><><><><><><><><><><><><>

	//Clean up
	Py_DECREF(p_m1);
	Py_DECREF(p_m2);
	Py_DECREF(p_x);

	Py_INCREF(Py_None);
	return Py_None;

}