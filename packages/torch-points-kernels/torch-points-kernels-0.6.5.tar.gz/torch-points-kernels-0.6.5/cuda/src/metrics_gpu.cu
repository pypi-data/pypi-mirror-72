#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include "cuda_utils.h"

#define THREADS 512

__global__ void instance_iou_cuda_kernel(
    long total_gt_instances, const long* __restrict__ nInstance, int nProposal,
    const long* __restrict__ proposals_idx, const long* __restrict__ proposals_offset,
    const long* __restrict__ instance_labels, const long* __restrict__ offset_num_gt_instances,
    const long* __restrict__ batch, const long* __restrict__ instance_pointnum,
    float* proposals_iou)
{
    for (int proposal_id = blockIdx.x; proposal_id < nProposal; proposal_id += gridDim.x)
    {
        int start = proposals_offset[proposal_id];
        int end = proposals_offset[proposal_id + 1];
        int sampleIdx = batch[proposals_idx[start]];
        int sampleNInstances = nInstance[sampleIdx];
        int instanceOffset = offset_num_gt_instances[sampleIdx];
        int proposal_total = end - start;
        for (int instance_id = threadIdx.x; instance_id < sampleNInstances;
             instance_id += blockDim.x)
        {
            int instance_total = instance_pointnum[instanceOffset + instance_id];
            int intersection = 0;
            for (int i = start; i < end; i++)
            {
                int idx = proposals_idx[i];
                if ((int)instance_labels[idx] == instance_id + 1)
                { // 0 is reserved for "no instance"
                    intersection += 1;
                }
            }

            proposals_iou[instanceOffset + instance_id + proposal_id * total_gt_instances] =
                (float)intersection /
                ((float)(proposal_total + instance_total - intersection) + 1e-5);
        }
    }
}

// input: proposals_idx (sumNPoint), int
// input: proposals_offset (nProposal + 1), int
// input: instance_labels (N), long, 0~total_nInst-1, -100
// input: instance_pointnum (total_nInst), int
// output: proposals_iou (nProposal, total_nInst), float
void instance_iou_kernel_wrapper(long total_gt_instances, long max_gt_instances,
                                 const long* nInstance, int nProposal, const long* proposals_idx,
                                 const long* proposals_offset, const long* instance_labels,
                                 const long* offset_num_gt_instances, const long* batch,
                                 const long* instance_pointnum, float* proposals_iou)
{
    auto stream = at::cuda::getCurrentCUDAStream();
    instance_iou_cuda_kernel<<<std::min(nProposal, THREADS * THREADS),
                               std::min(max_gt_instances, (long)THREADS), 0, stream>>>(
        total_gt_instances, nInstance, nProposal, proposals_idx, proposals_offset, instance_labels,
        offset_num_gt_instances, batch, instance_pointnum, proposals_iou);
}
